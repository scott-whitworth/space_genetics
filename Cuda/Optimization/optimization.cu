#include "hip/hip_runtime.h"
// Didymos Optimization Project using CUDA and a genetic algorithm

//TODO: Clarify complexities of the include paths
//TODO: What / why we are including
#include "../Earth_calculations/earthInfo.h"  // For launchCon and EarthInfo()
#include "../Genetic_Algorithm/individuals.h" // For individual structs, paths to rkParameters for randomParameters()
#include "../Output_Funcs/output.h" // For terminalDisplay(), recordGenerationPerformance(), and finalRecord()
#include "../Runge_Kutta/runge_kuttaCUDA.cuh" // for testing rk4simple
#include "../Genetic_Algorithm/ga_crossover.h" // for selectSurvivors() and newGeneration()

#include <iostream> // cout
#include <iomanip>  // used for setw(), sets spaces between values output
#include <random>   // for std::mt19937_64 object
#include <vector>
// Used to see if the best individual is changing when compared to a previous individual across generations 
// Returns true if the currentBest is not equal to previousBest within a distinguishable difference
// Input: previousBestPos - Position diference at the end of RK simulation, in AU
//        previousBestVel - Velocity difference, in AU/s (currently not implemented)
//        currentBest     - 'best' individual from current run, based on how individuals are sorted
//        distinguishRate - magnitude of difference
// Called inside of optimize to see if anneal rate needs to change

//! Currently not in use, changeInBest determined by cost, not posDiff
// bool changeInBest(double previousBestPos, double previousBestVel, const Individual & currentBest, double distinguishRate) {
//     //truncate is used here to compare doubles via the distinguguishRate, to ensure that there has been relatively no change.
//     if (trunc(previousBestPos/distinguishRate) != trunc(currentBest.posDiff/distinguishRate)) {
//         return true;
//     }
//     else {
//         /* //Used if Velocity should be considered
//         if (trunc(previousBestVel/distinguishRate) != trunc(currentBest.speedDiff/distinguishRate)) {
//             return true;
//         }
//         else return false;
//         */
//         return false;
//     }
// }


//----------------------------------------------------------------------------------------------------------------------------
//Returns the number of fronts and the number of individuals per front
void countFrontSize(std::vector<int> &frontCounter, const cudaConstants* cConstants, int generation, Individual* pool, Individual* parentPool, const int parentPoolSize) {
    //reallocate space for frontCounter. Delete last generation's frontCounter first
    
    //Reset the vector to be the correct size (the size of the vector is equivalent to the number of fronts)
    std::vector<int>().swap(frontCounter);
    frontCounter.resize(pool[parentPoolSize*2 - 1].rank);


    //Fill frontCounter.
    for (int i = 0; i < parentPoolSize*2; i++) {
        if (pool[i].rank <= 0) {
            std::cout << "Error Detected In FillParentPool!" << std::endl;
        }
        else {
            frontCounter[pool[i].rank - 1] += 1;
        }
    }

}



//----------------------------------------------------------------------------------------------------------------------------
//Used to give rankings for sorting based on non-dominated sorting method.
//Assigns suitability rank to all individuals.
//MUST be called after cost has been assigned to all individuals (calling callRK)
//Input: pool - this generation of individuals, defined/initilized in optimimize
//       cConstants
void giveRank(Individual * pool, const cudaConstants* cConstants) {
    //non-denominated sorting method attempt
    //https://www.iitk.ac.in/kangal/Deb_NSGA-II.pdf

    //Used to store the current front of individuals. first filled with the first front individuals(best out of all population)
    // filled with index of individuals in pool
    std::vector<int> front;
    
    //loop through each individual
    for (int i = 0; i < cConstants->num_individuals*2; i++){
        
        //number of times pool[i] has been dominated
        pool[i].dominatedCount = 0;

        //set of solutions that pool[i] dominates. Need to empty for each generation
        std::vector<int>().swap(pool[i].dominated);

        for(int j = 0; j < cConstants->num_individuals*2; j++){
            
            //if i dominates j, put the j index in the set of individuals dominated by i.
            if (dominates(pool[i], pool[j])){
                pool[i].dominated.push_back(j);
            }
            //if j dominates i, increase the number of times that i has been dominated
            else if (dominates(pool[j], pool[i])) {
                pool[i].dominatedCount++;
            }
        }
        
        //if i was never dominated, add it's index to the best front, front1. Making its ranking = 1.
        if (pool[i].dominatedCount == 0){
            pool[i].rank = 1;
            front.push_back(i);
        }
        //std::cout << i << " ";
    }

    //Used to assign rank number
    int rankNum = 1;
    //vector to store individuals' indexes in next front
    std::vector<int> newFront;

    //go until all individuals have been put in better ranks and none are left to give a ranking
    while(!front.empty()) {
        //empty the new front to put new individuals in
        std::vector<int>().swap(newFront);

        //loop through all individuals in old front
        for(int k = 0; k < front.size(); k++){

            //loop through all the individuals that the individual at index k dominated
            for(int l = 0; l < pool[front[k]].dominated.size(); l++){

                //subtract 1 from the dominated individuals' dominatedCount.
                //if an individual was dominated only once for example, it would be on the second front of individuals.
                pool[pool[front[k]].dominated[l]].dominatedCount--;

                //if the dominated count is at 0, add the individual to the next front and make its rank equal to the next front number.
                if (pool[pool[front[k]].dominated[l]].dominatedCount == 0){
                    pool[pool[front[k]].dominated[l]].rank = rankNum + 1;
                    newFront.push_back(pool[front[k]].dominated[l]);                        
                }
            }
        }
        //increment the rank number
        rankNum++;

        std::vector<int>().swap(front);
        //go to next front
        front = newFront;
    }
    //for each individual, check if it is a NaN. If it is, give it a very low rank.
    // for (int i = 0; i < cConstants->num_individuals*2; i++){
    //     if (pool[i].cost > 99.9 && pool[i].cost < 100.01){
    //         pool[i].rank = cConstants->num_individuals*2;
    //     }
    // }
}

//----------------------------------------------------------------------------------------------------------------------------
void giveDistance(Individual * pool, const cudaConstants* cConstants, int poolSize){

    //starting rankSort to make sure nans are at the end of the array.
    std::sort(pool, pool + cConstants->num_individuals*2, rankSort);

    for (int i = 0; i < poolSize; i++ ){
        //reset each individual's distance
        pool[i].distance = 0.0;
    }

    std::sort(pool, pool + poolSize, LowerPosDiff);
    pool[0].distance = 1.0e+12;
    pool[poolSize - 1].distance = 1.0e+12;

    //For each individual besides the upper and lower bounds, make their distance equal to
    //the current distance + the absolute normalized difference in the function values of two adjacent solutions.
    double normalPosDiffLeft;
    double normalPosDiffRight;
    for(int i = 1; i < poolSize - 1; i++){
        //distance = distance + ((i+1) - (i-1))/(best - worst)
        normalPosDiffLeft = pool[i+1].posDiff/pool[poolSize - 1].posDiff;
        normalPosDiffRight = pool[i-1].posDiff/pool[poolSize - 1].posDiff;
        pool[i].distance = pool[i].distance + abs((normalPosDiffLeft - normalPosDiffRight));// /(pool[poolSize - 1].posDiff - pool[0].posDiff));
    }

    //Repeat above process for speedDiff    
    std::sort(pool, pool + poolSize, LowerSpeedDiff);
    pool[0].distance = 1.0e+12;
    pool[poolSize - 1].distance = 1.0e+12;
    double normalSpeedDiffLeft;
    double normalSpeedDiffRight;
    //For each individual besides the upper and lower bounds, make their distance equal to
    //the current distance + the absolute normalized difference in the function values of two adjacent solutions.
    for(int i = 1; i < poolSize - 1; i++){
        normalSpeedDiffLeft = pool[i+1].speedDiff/pool[poolSize - 1].speedDiff;
        normalSpeedDiffRight = pool[i-1].speedDiff/pool[poolSize - 1].speedDiff;
        pool[i].distance = pool[i].distance + abs((normalSpeedDiffLeft - normalSpeedDiffRight));// /(pool[poolSize - 1].speedDiff - pool[0].speedDiff));
    }
    // double tolerance = 1.0e-11;
    // for(int i = 0; i < poolSize; i++){
    //     if((pool[i].cost < pool[0].cost + tolerance) && (pool[i].cost > pool[0].cost - tolerance)){
    //         pool[i].distance = 0;
    //     }
    //     else {
    //         pool[i].distance = 10;
    //     }
    // }
}

//----------------------------------------------------------------------------------------------------------------------------
void fillParentPool(Individual * entirePool, Individual * parentPool, const cudaConstants* cConstants, int entirePoolSize){
    //sort all individuals based on rank
    std::sort(entirePool, entirePool + entirePoolSize, rankSort);         

    //the current rank of an individual
    int currentRank = 1;
    //the current index of an individual
    int currentIndex = 0;
    //the index of the first individual with the current rank
    int currentRankFirstIndex;
    //the index of the last individual with the current rank
    int currentRankLastIndex;
    int nextRank = 2;
    //the sum of all individuals counted so far
    int totalAdded = 0;

    //while number of individuals added to new parent population is less than how many we want 
    while(totalAdded < entirePoolSize/2){

        //how many individuals have a certain rank
        int rankCount = 0;
        //the index of the first individual in a rank
        currentRankFirstIndex = currentIndex;
        //while the rank has not gone to the next rank, added another individual to rankCount
        while(currentRank < nextRank) {
            rankCount++;
            //goes until it finds the idnividual with the next rank
            currentIndex++;
            currentRank = entirePool[currentIndex].rank;   
        }
        //the index of the last individual in a rank
        currentRankLastIndex = currentIndex - 1;
        totalAdded += rankCount;
        nextRank++;
    }
    //sort the individuals in last rank to be added by the distance.
    std::sort(entirePool + currentRankFirstIndex, entirePool + currentRankLastIndex, rankDistanceSort);
    //loop through individuals and fill new parent array with best individuals.
    for(int i = 0; i < entirePoolSize/2; i++){
        parentPool[i] = entirePool[i];
    }
}

//----------------------------------------------------------------------------------------------------------------------------
bool changeInBest(double previousBestCost, const Individual & currentBest, double distinguishRate) {
    //truncate is used here to compare doubles via the distinguguishRate, to ensure that there has been relatively no change.
        if (trunc(previousBestCost/distinguishRate) != trunc(currentBest.cost/distinguishRate)) {
            return true;
        }
        else { 
            return false;
        }
}

//----------------------------------------------------------------------------------------------------------------------------
// ** Assumes pool is sorted array of Individuals **
// Used in determining if main optimize loop continues
// Input: tolerance - posDiff threshold, determines max target distance
//        pool - this generation of Individuals, defined/initilized in optimimize
//        cConstants - struct holding config values, used for accessing best_count value
// Output: Returns true if top best_count individuals within the pool are within the tolerance
bool allWithinTolerance(double tolerance, Individual * pool, const cudaConstants* cConstants) {

    //Check what type of mission is running to use the correct cost function
    if (cConstants->missionType == Rendezvous){
        // Iterate to check best_count number of 'top' individuals
        for (int i = 0; i < cConstants->best_count; i++) {
            // if(pool[i].getCost_Soft(cConstants) >= tolerance) {
            //     //One was not within tolerance
            //     return false;
            // }
            if (pool[i].posDiff >= tolerance){
                return false;
            }

            if (pool[i].speedDiff >= tolerance){
                return false;
            }
        }
    }
    else if(cConstants->missionType == Impact){
        // Iterate to check best_count number of 'top' individuals
        for (int i = 0; i < cConstants->best_count; i++) {
            
            if(pool[i].getCost_Hard(cConstants) >= tolerance) {
                //One was not within 
                return false;
            }
        }  
    }

    // If iterated through and all were within tolerance, success
    return true;
}

//----------------------------------------------------------------------------------------------------------------------------
// Main processing function for Genetic Algorithm
// - manages memory needs for genetic algorithm
// - deals with processing calls to CUDA callRK
// - exits when individuals converge on tolerance defined in Constants
double optimize(const cudaConstants* cConstants) {
    // Not used, previously used for reporting computational performance
    double calcPerS = 0;

    time_t timeSeed = cConstants->time_seed;
    std::mt19937_64 rng(timeSeed); // This rng object is used for generating all random numbers in the genetic algorithm, passed in to functions that need it
    
    std::cout << "----------------------------------------------------------------------------------------------------" << std::endl;
       
    // Initialize the recording files if in record mode
    if (cConstants->record_mode == true) {
        initializeRecord(cConstants);
    }
     
    // input parameters for Runge Kutta process
    // Each parameter is the same for each thread on the GPU
    double timeInitial = 0; // the starting time of the trip is always defined as zero   
    // Runge Kutta adaptive time step error tolerance
    double absTol = cConstants->rk_tol; 
    // the starting step size for RK run
    // - note that the current step size varies throughout each run
    //TODO: Should this be based on max_numsteps?
    double stepSize = ((cConstants->orbitalPeriod) - timeInitial) / cConstants->GuessMaxPossibleSteps; 

    // Initial genetic anneal scalar
    double currentAnneal = cConstants->anneal_initial;

    // Main set of parameters for Genetic Algorithm
    // contains all thread unique input parameters
    Individual *inputParameters = new Individual[cConstants->num_individuals]; 

    // Main set of parameters for Genetic Algorithm, the old generation parameters
    // contains all thread unique input parameters
    Individual *oldInputParameters = new Individual[cConstants->num_individuals]; 

    //the set of all old and new individuals
    Individual *allIndividuals = new Individual[cConstants->num_individuals*2];

    // set to zero to force difference in first generation
    // double previousBestPos = 0; 
    // double previousBestVel = 0;
    double previousBestCost = 0;

    // Initilize individuals randomly or from a file
    if (cConstants->random_start) {
        // individuals set to randomly generated, but reasonable, parameters
        for (int i = 0; i < cConstants->num_individuals; i++) { 
            inputParameters[i] = Individual(randomParameters(rng, cConstants), cConstants);
        }
    }
    // Read from file using cConstants initial_start_file_address to get path
    else {
        // **Might be depreciated, not tested summer 2020**
        // Sets inputParameters to hold initial individuals based from file optimizedVector.bin
        const int numStarts = 14; // the number of different sets of starting parameters in the input file
        std::ifstream starts;
        starts.open(cConstants->initial_start_file_address, std::ifstream::in|std::ios::binary); // a file containing the final parameters of converged results from CPU calculations        

        // sort the data into 2 dimensions
        // one row is one set of starting parameters
        // each column is a specific variable:
        double startDoubles;
        // arrayCPU needs to be updated to handle the fact that OPTIM_VARS may be flexible
        double arrayCPU[numStarts][OPTIM_VARS];
        
        for (int i = 0; i < OPTIM_VARS; i++) { // rows
            for (int j = 0; j < numStarts; j++) { // columns
                starts.read( reinterpret_cast<char*>( &startDoubles ), sizeof startDoubles );
                arrayCPU[j][i] = startDoubles;
            }
        }
        starts.close();

         // set every thread's input parameters to a set of final values from CPU calculations for use as a good starting point
        for (int i = 0; i < cConstants->num_individuals; i++) {
            int row = rng() % numStarts; // Choose a random row to get the parameters from

            double tripTime = arrayCPU[row][TRIPTIME_OFFSET];
            double alpha = arrayCPU[row][ALPHA_OFFSET];
            double beta = arrayCPU[row][BETA_OFFSET];
            double zeta = arrayCPU[row][ZETA_OFFSET];

            coefficients<double> testcoeff;
            for (int j = 0; j < testcoeff.gammaSize; j++) {
                testcoeff.gamma[j] = arrayCPU[row][j + GAMMA_OFFSET];
            }

            for (int j = 0; j < testcoeff.tauSize; j++) {
                testcoeff.tau[j] =  arrayCPU[row][j + TAU_OFFSET];
            }

            for (int j = 0; j < testcoeff.coastSize; j++) {
                testcoeff.coast[j] = arrayCPU[row][j + COAST_OFFSET];
            }

            rkParameters<double> example(tripTime, alpha, beta, zeta, testcoeff); 

            inputParameters[i] = Individual(example, cConstants);
        }
    }

    // Collection of individuals used in the genetic selection process
    //  - filled in selectSurvivors, based on callRK output
    //  - stores the winners of the head-to-head competition
    Individual *survivors = new Individual[cConstants->survivor_count]; 

    // Number of individuals that need to be evaluated
    // - the whole population is in first loop
    // - subsequent generations only calculate *new* individuals
    int newInd = cConstants->num_individuals;

    // number of current generation
    double generation = 0;    
    
    // Genetic solution tolerance 
    // - (currently just the position threshold which is furthest distance from the target allowed)
    // - could eventually take into account velocity too and become a more complex calculation
    double tolerance = cConstants->pos_threshold; 
             
    // distinguishable rate used in changeInBest()
    //  - used to help check for a change in anneal
    //  - Gets smaller when no change is detected
    double dRate = 1.0e-8;

    // Flag for finishing the genetic process
    // set by allWithinTolerance()
    bool convergence = false;

    //Decrease the change check as time goes on without an anneal
    //int check_decrease = 1;


    //Each member is a counter for a front. The value of a member equals the number of individuals in that front. The size of the whole vector equals the number of fronts.
    std::vector<int> frontCounter(cConstants->num_individuals);

    // main gentic algorithm loop
    // - continues until allWithinTolerance returns true (specific number of individuals are within threshold)
    do {
        // each inputParameter represents an individual set of starting parameters
        // GPU based runge kutta process determines final position and velocity based on parameters
        // newInd - how many individuals that are *new* that need to be evaluated
        //        - All individuals first generation
        //        - only new individuals, from crossover, in subsequent generations
        // (inputParameters + (cConstants->num_individuals - newInd)) value accesses the start of the section of the inputParameters array that contains new individuals
        callRK(newInd, cConstants->thread_block_size, inputParameters + (cConstants->num_individuals - newInd), timeInitial, stepSize, absTol, calcPerS, cConstants); // calculate trajectories for new individuals

        //numNans - number of times a nan is found in 100 generations.
        int numNans = 0;


        // if we got bad results reset the Individual to random starting values (it may still be used for crossover) and set the final position to be way off so it gets replaced by a new Individual
        for (int k = 0; k < cConstants->num_individuals; k++) {
            //Checking each individuals final position for NaNs
            if (isnan(inputParameters[k].finalPos.r) || isnan(inputParameters[k].finalPos.theta) || isnan(inputParameters[k].finalPos.z) || isnan(inputParameters[k].finalPos.vr) || isnan(inputParameters[k].finalPos.vtheta) || isnan(inputParameters[k].finalPos.vz)) {
                //std::cout << std::endl << std::endl << "NAN FOUND" << std::endl << std::endl;
                numNans++;
                //inputParameters[k] = Individual(randomParameters(rng, cConstants), cConstants);
                // Set to be a bad individual by giving it bad posDiff and speedDiffs
                // therefore also having a bad cost value
                // won't be promoted in crossover
                inputParameters[k].posDiff = 100.0;//This is an undesirable position difference of 100 AU

                if (cConstants->missionType == Rendezvous){
                    inputParameters[k].speedDiff = 100.0;//This is an undesirable result for an rendezvous mission (approx. 50000c!)
                    // calculate its new cost function based on 'bad' differences
                    inputParameters[k].cost = 100.0;
                }
                else if (cConstants->missionType == Impact){
                    inputParameters[k].speedDiff = 0.0;//This is an undesirable result for an impact mission
                    // calculate its new cost function based on 'bad' differences
                    inputParameters[k].cost = 100.0;                   
                }
            }
            
        }
        if (generation == 0) {
            numNans *= 2;
        }


        //fill with new individuals
        for(int i = 0; i < cConstants->num_individuals; i++){
            allIndividuals[i] = inputParameters[i];
        }
        //fill with old individuals
        if(generation == 0){
            for(int i = 0; i < cConstants->num_individuals; i++){
                allIndividuals[i + cConstants->num_individuals] = inputParameters[i];
            }
        } 
        else {
            for(int i = 0; i < cConstants->num_individuals; i++){
                allIndividuals[i + cConstants->num_individuals] = oldInputParameters[i];
            }
        }

        //* Push the nans to the back of the vector natural individuals have rank = 0; nans have rank = 2880.
        //std::sort(allIndividuals, allIndividuals + cConstants->num_individuals*2, rankSort);

        //give a rank to each individual based on domination sort
        //* Ignore any nans at the end of allIndividuals
        //must be called after checking for nans and before giveDistance
        giveRank(allIndividuals, cConstants);

        giveDistance(allIndividuals, cConstants, cConstants->num_individuals*2 - numNans);
        std::sort(allIndividuals, allIndividuals + cConstants->num_individuals * 2, rankDistanceSort);
        
        //sort by rank distance and then fill inputParameters with the best.
        
        //countFrontSize(frontCounter, cConstants, generation, allIndividuals, inputParameters, cConstants->num_individuals);

        // std::sort(allIndividuals, allIndividuals + cConstants->num_individuals*2, rankSort);
        for(int i = 0; i < cConstants->num_individuals; i++){
            inputParameters[i] = allIndividuals[i];
            //std::cout << "Rank: " << inputParameters[i].rank << " | Distance: " << inputParameters[i].distance << std::endl;
        }
        //fillParentPool(allIndividuals, inputParameters, cConstants, cConstants->num_individuals*2);
        //for(int i = 0; i < cConstants->num_individuals; i++){
            // std::cout << " " << allIndividuals[i].rank;
            // if(allIndividuals[i].rank != allIndividuals[i + 1].rank){
            //     std::cout << std::endl;
            // }
            //std::cout << "Rank: " << inputParameters[i].rank << " | Distance: " << inputParameters[i].distance << std::endl;
       // }
        // Preparing survivor pool with individuals for the newGeneration crossover
        // Survivor pool contains:
        //               - individuals with best PosDiff
        //               - individuals with best speedDiffs
        //               - depends on cConstants->sortingRatio (0.1 is 10% are best PosDiff for example)
        // inputParameters is left sorted by individuals with best speedDiffs 
        selectSurvivors(inputParameters, cConstants->num_individuals, cConstants->survivor_count, survivors, cConstants->sortingRatio, cConstants->missionType); // Choose which individuals are in survivors, current method selects half to be best posDiff and other half to be best speedDiff

        if (static_cast<int>(generation) % cConstants->disp_freq == 0) {
            recordAllIndividuals("Survivors", cConstants, survivors, cConstants->survivor_count, generation);
        }

        //std::sort(inputParameters, inputParameters + cConstants->num_individuals, rankSort);
        //fillParentPool(allIndividuals, inputParameters, cConstants, cConstants->num_individuals*2);
        //std::sort(inputParameters, inputParameters + cConstants->num_individuals, rankSort);
        // Display a '.' to the terminal to show that a generation has been performed
        // This also serves to visually seperate the terminalDisplay() calls across generations 
        std::cout << '.';

        //sort new parent individuals by cost so that we can check the first individual for change
        if (static_cast<int>(generation) % (cConstants->change_check) == 0) {
            std::sort(inputParameters, inputParameters + cConstants->num_individuals);
        }
        else {
            std::sort(inputParameters, inputParameters + cConstants->num_individuals, rankDistanceSort);
        }

        // std::sort(inputParameters, inputParameters + cConstants->num_individuals, rankSort);
        // Calculate how far best individual is from the ideal cost value (currently is the positionalDifference of the best individual)
        // TODO: Change this later to take into account more than just the best individual and its position difference
        // how far away the best individual is from the tolerance value
        //double currentCost; 
        //currentCost = inputParameters[0].cost; 

        // Scaling anneal based on proximity to tolerance
        // Far away: larger anneal scale, close: smaller anneal
        double new_anneal;
        if (tolerance < inputParameters[0].posDiff){    
            //new_anneal = currentAnneal * (1 - (tolerance / inputParameters[0].posDiff));
            new_anneal = currentAnneal * (1 - pow(tolerance / inputParameters[0].posDiff,2.0));
            if (new_anneal<1.0e-7){
                new_anneal = 1.0e-7;//Set a true minimum for annealing
            }
        }
        //double new_anneal = cConstants->anneal_initial * (1 - tolerance / inputParameters[0].posDiff);

        //Process to see if anneal needs to be adjusted
        // If generations are stale, anneal drops
        Individual currentBest;
        // Compare current best individual to that from CHANGE_CHECK many generations ago.
        // If they are the same, change size of mutations
        if (static_cast<int>(generation) % (cConstants->change_check) == 0) { 
            currentBest = inputParameters[0];
            // checks for anneal to change
            // previousBest starts at 0 to ensure changeInBest = true on generation 0
            if ( !(changeInBest(previousBestCost, currentBest, dRate)) ) { 
                //this ensures that changeInBest never compares two zeros, thus keeping dRate in relevance as the posDiff lowers
                if (trunc(currentBest.cost/dRate) == 0) { 
                    while (trunc(currentBest.cost/dRate) == 0) {
                        dRate = dRate/10; 
                    }
                    std::cout << "\nnew dRate: " << dRate << std::endl;
                }
                // If no change in BestIndividual across generations, reduce currentAnneal by anneal_factor while staying above anneal_min
//                double anneal_min = cConstants->anneal_initial*exp(-sqrt(tolerance/inputParameters[0].posDiff)*generation);
                double anneal_min = cConstants->anneal_initial*exp(-sqrt(tolerance/inputParameters[0].posDiff)*generation);
                if (anneal_min<1.0e-7){
                    anneal_min = 1.0e-7;//Set a true minimum for annealing
                }
                currentAnneal = (currentAnneal * cConstants->anneal_factor > anneal_min)? (currentAnneal * cConstants->anneal_factor):(anneal_min);
                std::cout << "\nnew anneal: " << currentAnneal << std::endl;              
            }
            // previousBestPos = currentBest.posDiff;
            // previousBestVel = currentBest.speedDiff;
            previousBestCost = currentBest.cost;
        }

        // If in recording mode and write_freq reached, call the record method
        if (static_cast<int>(generation) % cConstants->write_freq == 0 && cConstants->record_mode == true) {
            recordGenerationPerformance(cConstants, inputParameters, generation, new_anneal, cConstants->num_individuals, frontCounter.size());
        }

        // Only call terminalDisplay every DISP_FREQ, not every single generation
        if ( static_cast<int>(generation) % cConstants->disp_freq == 0) {
            // Prints the best individual's posDiff / speedDiff and cost
            //terminalDisplay(inputParameters[0], generation);

            //best position individual
            std::cout << "\nBest Position Individual: \n";
            std::sort(inputParameters, inputParameters + cConstants->num_individuals, LowerPosDiff);
            terminalDisplay(inputParameters[0], generation);

            if(cConstants->missionType == Rendezvous){
                //Best lower speed individual
                std::cout << "\nBest Speed Individual: \n";
                std::sort(inputParameters, inputParameters + cConstants->num_individuals, LowerSpeedDiff);
                terminalDisplay(inputParameters[0], generation);
            }
            else if(cConstants->missionType == Impact){
                //Best higher speed individual
                std::cout << "\nBest Speed Individual: \n";
                std::sort(inputParameters, inputParameters + cConstants->num_individuals, HigherSpeedDiff);
                terminalDisplay(inputParameters[0], generation);
            }

            std::sort(inputParameters, inputParameters + cConstants->num_individuals);
            terminalDisplay(inputParameters[0], generation);
            std::cout << "\n# of Nans this generation: " << numNans << "\n" << std::endl;
            

            std::sort(inputParameters, inputParameters+cConstants->num_individuals, rankDistanceSort);
            recordAllIndividuals("AllIndividuals-End", cConstants, inputParameters, cConstants->num_individuals, generation);
            // std::sort(inputParameters, inputParameters+cConstants->num_individuals, rankSort);
            //recordAllIndividuals(cConstants, inputParameters, cConstants->num_individuals, generation);

            
            //Reset the tally of nans.
            numNans = 0;
            
        }

        // Before replacing new individuals, determine whether all are within tolerance
        // Determines when loop is finished
        convergence = allWithinTolerance(tolerance, inputParameters, cConstants);


        //store away the old individuals
        for (int i = 0; i < cConstants->num_individuals; i++){
            oldInputParameters[i] = inputParameters[i];
        }
        //oldInputParameters = inputParameters;

        // Create a new generation and increment the generation counter
        // Genetic Crossover and mutation occur here
        newInd = newGeneration(survivors, inputParameters, cConstants->survivor_count, cConstants->num_individuals, new_anneal, cConstants, rng, generation);
        ++generation;
    
        //Loop exits based on result of allWithinTolerance and if max_generations has been hit
    } while ( !convergence && generation < cConstants->max_generations);

    // Call record for final generation regardless of frequency
    // for the annealing argument, set to -1 (since the anneal is only relevant to the next generation and so means nothing for the last one)
    // for the numFront argument, set to -1 (just because)
    if (cConstants->record_mode == true) {
        recordGenerationPerformance(cConstants, oldInputParameters, generation, currentAnneal, cConstants->num_individuals, -1);
    }
    // Only call finalRecord if the results actually converged on a solution
    // also display last generation onto terminal
    if (convergence) {
        terminalDisplay(oldInputParameters[0], generation);
        finalRecord(cConstants, oldInputParameters, static_cast<int>(generation));
    }
    
    delete [] inputParameters;
    delete [] survivors;
    delete [] oldInputParameters;
    delete [] allIndividuals;

    return calcPerS;
}
//----------------------------------------------------------------------------------------------------------------------------
int main () {
    // display GPU properties and ensure we are using the right one
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    std::cout << "\n\nDevice Number: 0 \n";
    std::cout << "- Device name: " << prop.name << std::endl << std::endl;
    hipSetDevice(0);
    
    // Declare the genetic constants used, with file path being used to receive initial values
    cudaConstants * cConstants = new cudaConstants("../Config_Constants/genetic.config"); 

    // Sets run0 seed, used to change seed between runs
    // Seed is set in cudaConstants: current time or passed in via config
    double zero_seed = cConstants->time_seed;
    // Perform the optimization with optimize function
    for (int run = 0; run < cConstants->run_count; run++) {
        // Adjust the time_seed so it is unique based on each run
        cConstants->time_seed = zero_seed + run*100;

        // Display contents of cConstants being used for this run and how many runs
        std::cout << *cConstants;
        std::cout << "\tPerforming run #" << run+1 << "\n\n";

        // pre-calculate a table of Earth's position within possible mission time range
        // defined as global variable
        // accessed on the CPU when individuals are initilized
        launchCon = new EarthInfo(cConstants); 

        // File output of element values that were calculated in EarthInfo constructor for verification
        /*if (cConstants->record_mode == true) {
            recordEarthData(cConstants, run);
        }*/
        // Call optimize with the current parameters in cConstants
        optimize(cConstants);

        delete launchCon; // Deallocate launchCon info for this run as it may be using a different time range in the next run
    }
    // Now that the optimize function is done (assumed that optimize() also records it), deallocate memory of the cudaConstants
    delete cConstants;
    
    return 0;
}
