#include "hip/hip_runtime.h"
#include <math.h>
#include <iostream>
#include <fstream> // for outputing to .csv file
#include <chrono>
#include <algorithm> // sort(), shuffle()
#include <random>

// Called by optimize() in optimization.cu
void callRK(const int numThreads, const int blockThreads, Child *generation, double timeInitial, double stepSize, double absTol, double & calcPerS, const cudaConstants* cConstant) {
    
    hipEvent_t kernelStart, kernelEnd;
    hipEventCreate(&kernelStart);
    hipEventCreate(&kernelEnd);

    Child *devGeneration; 
    double *devTimeInitial;
    double *devStepSize;
    double *devAbsTol;
    cudaConstants *devCConstant;

    // allocate memory for the parameters passed to the device
    hipMalloc((void**) &devGeneration, numThreads * sizeof(Child));
    hipMalloc((void**) &devTimeInitial, sizeof(double));
    hipMalloc((void**) &devStepSize, sizeof(double));
    hipMalloc((void**) &devAbsTol, sizeof(double));
    hipMalloc((void**) &devCConstant, sizeof(cudaConstants));

    // copy values of parameters passed from host onto device
    hipMemcpy(devGeneration, generation, numThreads * sizeof(Child), hipMemcpyHostToDevice);
    hipMemcpy(devTimeInitial, &timeInitial, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(devStepSize, &stepSize, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(devAbsTol, &absTol, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(devCConstant, cConstant, sizeof(cudaConstants), hipMemcpyHostToDevice);
    

    // GPU version of rk4Simple()
    hipEventRecord(kernelStart);
    rk4SimpleCUDA<<<(numThreads+blockThreads-1)/blockThreads,blockThreads>>>(devGeneration, devTimeInitial, devStepSize, devAbsTol, numThreads, devCConstant);
    hipEventRecord(kernelEnd);

    // copy the result of the kernel onto the host
    hipMemcpy(generation, devGeneration, numThreads * sizeof(Child), hipMemcpyDeviceToHost);
    
    // free memory from device
    hipFree(devGeneration);
    hipFree(devTimeInitial);
    hipFree(devStepSize);
    hipFree(devAbsTol);
    hipFree(devCConstant);

    float kernelT;
    
    hipEventSynchronize(kernelEnd);

    hipEventElapsedTime(&kernelT, kernelStart, kernelEnd);
    
    calcPerS = numThreads / (kernelT / 1000.0); // how many times the Runge Kutta algorithm ran in the kernel per second
}

// seperate conditions are passed for each thread, but timeInitial, stepSize, and absTol are the same for every thread
__global__ void rk4SimpleCUDA(Child *children, double *timeInitial, double *startStepSize, double *absTolInput, int n, const cudaConstants* cConstant) {
    int threadId = threadIdx.x + blockIdx.x * blockDim.x;
    if (threadId < n) {
        rkParameters<double> threadRKParameters = children[threadId].startParams; // get the parameters for this thread

        elements<double> curPos = threadRKParameters.y0; // start with the initial conditions of the spacecraft

        // storing copies of the input values
        double stepSize = *startStepSize;
        double absTol = *absTolInput;
        double curTime = *timeInitial;
        double startTime = *timeInitial;
        double curAccel = 0;

        thruster<double> thrust(cConstant);

        double massFuelSpent = 0; // mass of total fuel expended (kg) starts at 0

        bool coast; // to hold the result from calc_coast()

        elements<double> error; // holds output of previous value from rkCalc

        while (curTime < threadRKParameters.tripTime) {

            // Check the thruster type before performing calculations
            if (cConstant->thruster_type == thruster<double>::NO_THRUST) {
                coast = curAccel = 0;
            }
            else {
                coast = calc_coast(threadRKParameters.coeff, curTime, threadRKParameters.tripTime, thrust);
                curAccel = calc_accel(curPos.r, curPos.z, thrust, massFuelSpent, stepSize, coast, static_cast<double>(cConstant->wet_mass), cConstant);
            }

            // calculate k values and get new value of y
            rkCalc(curTime, threadRKParameters.tripTime, stepSize, curPos, threadRKParameters.coeff, curAccel, error); 

            curTime += stepSize; // update the current time in the simulation
            
            stepSize *= calc_scalingFactor(curPos-error,error,absTol, cConstant->doublePrecThresh); // Alter the step size for the next iteration

            // The step size cannot exceed the total time divided by 2 and cannot be smaller than the total time divided by 1000
            if (stepSize > (threadRKParameters.tripTime - startTime) / cConstant->min_numsteps) {
                stepSize = (threadRKParameters.tripTime - startTime) / cConstant->min_numsteps;
            }
            else if (stepSize < (threadRKParameters.tripTime - startTime) / cConstant->max_numsteps) {
                stepSize = (threadRKParameters.tripTime - startTime) / cConstant->max_numsteps;
            }
            
            if ( (curTime + stepSize) > threadRKParameters.tripTime) {
                stepSize = (threadRKParameters.tripTime - curTime); // shorten the last step to end exactly at time final
            }

            // if the spacecraft is within 0.5 au of the sun, the radial position of the spacecraft artificially increases to 1000, to force that path to not be used in the optimization.
            if ( sqrt(pow(curPos.r,2) + pow(curPos.z,2)) < cConstant->sun_r_min) {
                //This is a bad result, needs to be set to be removed
                //Setting the child's status to be a sun error
                children[threadId].status = SUN_ERROR;

                //Set the child's diffs to undesirable values
                //First, set posDiff to a high value to make the sorting algorithim think it ended up far from the asteroid
                children[threadId].posDiff = BAD_POSDIFF;

                //Set the velocity difference to a bad value
                //Different depending on the mission type, so we need to check and see what type of mission this is
                if (cConstant-> missionType == Impact) {
                    //Set a low speed diff so this individual is less likely to be selected for future generations
                    children[threadId].speedDiff = BAD_HARD_SPEEDDIFF; 
                }  
                else {
                    //Set a high speed diff, which is bad for rendezvous missions
                    //This makes it unlikely to be selected as a parent
                    children[threadId].speedDiff = BAD_SOFT_SPEEDDIFF;
                }

                return;
            }
        }

        //Setting the status of the child to be valid
        children[threadId].status = VALID; 
         // output to this thread's index
        children[threadId].finalPos = curPos;

        // Calculate new values for this thread
        children[threadId].getPosDiff(cConstant);
        children[threadId].getSpeedDiff(cConstant);
        return;
    }
    return;
}

