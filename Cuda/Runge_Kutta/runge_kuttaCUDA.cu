#include "hip/hip_runtime.h"
#include <math.h>
#include <iostream>
#include <fstream> // for outputing to .csv file
#include <chrono>
#include <algorithm> // sort(), shuffle()
#include <random>

// Called by optimize() in optimization.cu
void callRK(const int numThreads, const int blockThreads, Child *generation, double timeInitial, double stepSize, double absTol, double & calcPerS, const cudaConstants* cConstant) {
    
    hipEvent_t kernelStart, kernelEnd;
    hipEventCreate(&kernelStart);
    hipEventCreate(&kernelEnd);

    Child *devGeneration; 
    double *devTimeInitial;
    double *devStepSize;
    double *devAbsTol;
    cudaConstants *devCConstant;

    // allocate memory for the parameters passed to the device
    hipMalloc((void**) &devGeneration, numThreads * sizeof(Child));
    hipMalloc((void**) &devTimeInitial, sizeof(double));
    hipMalloc((void**) &devStepSize, sizeof(double));
    hipMalloc((void**) &devAbsTol, sizeof(double));
    hipMalloc((void**) &devCConstant, sizeof(cudaConstants));

    // copy values of parameters passed from host onto device
    hipMemcpy(devGeneration, generation, numThreads * sizeof(Child), hipMemcpyHostToDevice);
    hipMemcpy(devTimeInitial, &timeInitial, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(devStepSize, &stepSize, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(devAbsTol, &absTol, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(devCConstant, cConstant, sizeof(cudaConstants), hipMemcpyHostToDevice);
    
    //std::cout << "\n~~~~~~~~~~~~~~~~~~~~~~\nTEST CHILD (0th):\n\tStart Status: " << generation[0].errorStatus << 
    //             "\n\tStart r: " << generation[0].startParams.y0.r << 
    //             "\n\tStart TT: " << generation[0].startParams.tripTime <<
    //             "\n\tFinal r: " << generation[0].finalPos.r;
    //std::cout << "\nSun-min test: " << cConstant->sun_r_min; 

    // GPU version of rk4Simple()
    hipEventRecord(kernelStart);
    rk4SimpleCUDA<<<(numThreads+blockThreads-1)/blockThreads,blockThreads>>>(devGeneration, devTimeInitial, devStepSize, devAbsTol, numThreads, devCConstant);
    hipEventRecord(kernelEnd);

    //std::cout << "\n~~~~~~~~~~~~~~~~~~~~~~\nTEST CHILD (0th):\n\tMid Status: " << generation[0].errorStatus << 
    //             "\n\tStart r: " << generation[0].startParams.y0.r << 
    //             "\n\tStart TT: " << generation[0].startParams.tripTime <<
    //             "\n\tFinal r: " << generation[0].finalPos.r;

    // copy the result of the kernel onto the host
    hipMemcpy(generation, devGeneration, numThreads * sizeof(Child), hipMemcpyDeviceToHost);

    //std::cout << "\n~~~~~~~~~~~~~~~~~~~~~~\nTEST CHILD (0th):\n\tEnd Status: " << generation[0].errorStatus << 
    //             "\n\tStart r: " << generation[0].startParams.y0.r << 
    //             "\n\tStart TT: " << generation[0].startParams.tripTime <<
    //             "\n\tFinal r: " << generation[0].finalPos.r;
    
    // free memory from device
    hipFree(devGeneration);
    hipFree(devTimeInitial);
    hipFree(devStepSize);
    hipFree(devAbsTol);
    hipFree(devCConstant);

    float kernelT;
    
    hipEventSynchronize(kernelEnd);

    hipEventElapsedTime(&kernelT, kernelStart, kernelEnd);
    
    calcPerS = numThreads / (kernelT / 1000.0); // how many times the Runge Kutta algorithm ran in the kernel per second
}

// seperate conditions are passed for each thread, but timeInitial, stepSize, and absTol are the same for every thread
__global__ void rk4SimpleCUDA(Child *children, double *timeInitial, double *startStepSize, double *absTolInput, int n, const cudaConstants* cConstant) {
    int threadId = threadIdx.x + blockIdx.x * blockDim.x;
    if (threadId < n) {
        rkParameters<double> threadRKParameters = children[threadId].startParams; // get the parameters for this thread

        elements<double> curPos = threadRKParameters.y0; // start with the initial conditions of the spacecraft

        // storing copies of the input values
        double stepSize = *startStepSize;
        double absTol = *absTolInput;
        double curTime = *timeInitial;
        double startTime = *timeInitial;
        double curAccel = 0;

        thruster<double> thrust(cConstant);

        double massFuelSpent = 0; // mass of total fuel expended (kg) starts at 0

        bool coast; // to hold the result from calc_coast()

        elements<double> error; // holds output of previous value from rkCalc

        while (curTime < threadRKParameters.tripTime) {

            // Check the thruster type before performing calculations
            if (cConstant->thruster_type == thruster<double>::NO_THRUST) {
                coast = curAccel = 0;
            }
            else {
                coast = calc_coast(threadRKParameters.coeff, curTime, threadRKParameters.tripTime, thrust);
                curAccel = calc_accel(curPos.r, curPos.z, thrust, massFuelSpent, stepSize, coast, static_cast<double>(cConstant->wet_mass), cConstant);
            }

            // calculate k values and get new value of y
            rkCalc(curTime, threadRKParameters.tripTime, stepSize, curPos, threadRKParameters.coeff, curAccel, error); 

            curTime += stepSize; // update the current time in the simulation
            
            stepSize *= calc_scalingFactor(curPos-error,error,absTol, cConstant->doublePrecThresh); // Alter the step size for the next iteration

            // The step size cannot exceed the total time divided by 2 and cannot be smaller than the total time divided by 1000
            if (stepSize > (threadRKParameters.tripTime - startTime) / cConstant->min_numsteps) {
                stepSize = (threadRKParameters.tripTime - startTime) / cConstant->min_numsteps;
            }
            else if (stepSize < (threadRKParameters.tripTime - startTime) / cConstant->max_numsteps) {
                stepSize = (threadRKParameters.tripTime - startTime) / cConstant->max_numsteps;
            }
            
            if ( (curTime + stepSize) > threadRKParameters.tripTime) {
                stepSize = (threadRKParameters.tripTime - curTime); // shorten the last step to end exactly at time final
            }

            // if the spacecraft is within 0.5 au of the sun, the radial position of the spacecraft artificially increases to 1000, to force that path to not be used in the optimization.
            if ( sqrt(pow(curPos.r,2) + pow(curPos.z,2)) < cConstant->sun_r_min) { //maybe issue is with using pow? I doubt it, but we could always try curPos.r*curPos.r + curPos.z*curPos.z < sun_r_min*sun_r_min?
                //This is a bad result, needs to be set to be removed
                //Setting the child's status to be a sun error
                children[threadId].errorStatus = SUN_ERROR;//Are all the children's errorStatus set to SUN_ERROR?

                return;
        }
        //Give the child its final calculated position
        children[threadId].finalPos = curPos;

        //if it is not a SUN_ERROR then it is valid
        children[threadId].errorStatus = VALID;

        return;
    }
    return;
}

