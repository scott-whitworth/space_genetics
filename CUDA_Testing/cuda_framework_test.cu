//Simple test program to test to see if CUDA is up an running


#include <hip/hip_runtime.h>
#include <iostream>

using std::cout;
using std::endl;

const int ARRAY_SIZE = 1000;

__global__ void simpleKenrel(double *space){
    int threadId = threadIdx.x + blockIdx.x * blockDim.x;
    if(threadId < ARRAY_SIZE){
        space[threadId] *= 2;
    }
    return;
}

int main(){

    // display GPU properties and ensure we are using the right one
    hipDeviceProp_t prop;
    int numDevices;
    hipGetDeviceCount(&numDevices);
    cout << "Number of devices detected: " << numDevices << endl;

    for(int dev = 0; dev < numDevices; dev++ ){
        hipGetDeviceProperties(&prop, 0);
        cout << "\nDevice Number: " << dev << " \n";
        cout << "- Device name: " << prop.name << endl;
        cout << "- Device compute: " << prop.major << "." << prop.minor << endl << endl;
    }

    //Create some arrays to fill with data
    double *host_input;
    double *device_space;
    double *host_output;

    //Allocate Memory
    hipMalloc((void**) &device_space, ARRAY_SIZE * sizeof(double));
    host_input = new double[ARRAY_SIZE];
    host_output = new double[ARRAY_SIZE];

    //Filling input: start with 3s
    for(int i = 0; i < ARRAY_SIZE; i++ ){
        host_input[i] = 3;
        host_output[i] = -1; //Should be overwritten with device_space
    }

    //Copy memory from host to device
    hipMemcpy(device_space, host_input, ARRAY_SIZE * sizeof(double), hipMemcpyHostToDevice);

    simpleKenrel<<<(ARRAY_SIZE+32-1)/32,32>>>(device_space);

    //Copy things off of device
    hipMemcpy(host_output, device_space, ARRAY_SIZE * sizeof(double), hipMemcpyDeviceToHost);

    bool check = true;
    for(int i = 0; i < ARRAY_SIZE; i++){
        //Visual inspection:
        //cout << host_output[i] << " ";
        if(host_output[i] != 6){
            check = false;
        }
    }

    if(check){
        cout << "Output is good!" << endl;
    } else {
        cout << "Output is bad!" << endl;
    }

    cout << endl;



    return 0;
}