#include "gpuMem.cuh"

 void GPUMem::initialize(const cudaConstants* cConstants, const int & marsConSize, const elements<double>* marsLaunchCon){
    //all these initializations taken from (old) line 31 of genetic_algorithm.cpp in callRK
    absTol = cConstants->rk_tol;
    numThreads = cConstants->num_individuals;

    int maxSteps = ((cConstants->max_numsteps+1) * cConstants->maxSimNum)+1;
    
    // allocate memory for the parameters passed to the device
    hipMalloc((void**) &devGeneration, numThreads * sizeof(Child));  
    hipMalloc((void**) &devAbsTol, sizeof(double));
    hipMalloc((void**) &devCConstant, sizeof(cudaConstants));
    hipMalloc((void**) &devMarsLaunchCon, marsConSize * sizeof(double));

    hipMalloc((void**) &devTime_steps, maxSteps * sizeof(double));
    hipMalloc((void**) &devY_steps, maxSteps * sizeof(elements<double>));
    hipMalloc((void**) &devGamma_steps, maxSteps * sizeof(double));
    hipMalloc((void**) &devTau_steps, maxSteps * sizeof(double));
    hipMalloc((void**) &devAccel_steps, maxSteps * sizeof(double));
    hipMalloc((void**) &devFuel_steps, maxSteps * sizeof(double));

    // copy values of parameters passed from host onto device
    hipMemcpy(devAbsTol, &absTol, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(devCConstant, cConstants, sizeof(cudaConstants), hipMemcpyHostToDevice);
    hipMemcpy(devMarsLaunchCon, marsLaunchCon, marsConSize * sizeof(double), hipMemcpyHostToDevice);
}

//deallocate the memory on the GPU when the code is finished
void GPUMem::free(){
    hipFree(devGeneration);
    // hipFree(devTimeInitial);
    // hipFree(devStepSize);
    hipFree(devAbsTol);
    hipFree(devCConstant);
    hipFree(devMarsLaunchCon);
    hipFree(devTime_steps);
    hipFree(devY_steps);
    hipFree(devGamma_steps);
    hipFree(devTau_steps);
    hipFree(devAccel_steps);
    hipFree(devFuel_steps);
}