#include "hip/hip_runtime.h"
// Didymos Optimization Project using CUDA and a genetic algorithm

//TODO: Clarify complexities of the include paths
//TODO: What / why we are including
#include "../Planet_calculations/planetInfo.h"  // For launchCon and EarthInfo()
#include "../Genetic_Algorithm/adult.h" // For adult structs, paths to rkParameters for randomParameters()
#include "../Genetic_Algorithm/child.h" // For child structs, paths to rkParameters for randomParameters()
#include "../Output_Funcs/output.h" // For terminalDisplay(), recordGenerationPerformance(), and finalRecord()
#include "../Runge_Kutta/runge_kuttaCUDA.cuh" // for testing rk4simple
#include "../Genetic_Algorithm/ga_crossover.h" // for selectSurvivors() and newGeneration()
#include "../Genetic_Algorithm/genetic_algorithm.h" //For functions that set up new generations
#include "../Genetic_Algorithm/sort.h" //For functions that will allow for sorting of the adult arrays by giving them ranks and distances
#include "../Genetic_Algorithm/anneal.h" //For all the annealing functions
#include "../Runge_Kutta/gpuMem.cuh" // for initializing and deallocating memory

//#include "../Unit_Testing/testing_sorts.cpp"

#include <iostream> // cout
#include <iomanip>  // used for setw(), sets spaces between values output
#include <random>   // for std::mt19937_64 object
#include <vector>   // allows us to use vectors instead of just arrays
#include <string>


//----------------------------------------------------------------------------------------------------------------------------
// ** Assumes pool is sorted array of Adults **
// Used in determining if main optimize loop continues
// Input: oldAdults - this generation of Adults, defined/initialized in optimimize
//        cConstants - struct holding config values, used for accessing best_count value and objectives list
// Output: Returns true if top best_count adults within the pool are within the tolerance
bool checkTolerance(std::vector<Adult> & oldAdults, const cudaConstants* cConstants);

//----------------------------------------------------------------------------------------------------------------------------
// TEST / LIKELY TEMPORARY FUNCTION
// This function will find the minimum, maximum, and average distance, average pos and speed diffs, the number of duplicates,
//                         the avg age, and the avg and max birthdays of the individuals in allAdults, which will then be used for reporting
// 
// Inputs:  allAdults - array of adults that will be considered
//          objectives - the vector of this run's objectives
//          objectiveAvgValues - a vector which will hold the generations's average parameter values for each of the objectives
//          duplicateNum - the number of duplicate adults found
//          minDist - minimum distance that will be calculated
//          avgDist - the average distance that will be calculated
//          maxDist - the maximum distance that will be calculated
//          generation - the current generation
//          avgAge  - the avegrage age of the adults, relative to the current generation
//          avgBirthday - average birth generation for the adults
//          oldestBirthday - the oldest adult's birth generation
// Outputs: The arguments will be filled in with the up-to-date values for this generation
void calculateGenerationValues (const std::vector<Adult> & allAdults, const std::vector<objective> & objectives, std::vector<double> & objectiveAvgValues, int & duplicateNum, double & minDist, double & avgDist, double & maxDist, const int & generation, double & avgAge, double & avgBirthday, int & oldestBirthday);

//----------------------------------------------------------------------------------------------------------------------------
// Main processing function for Genetic Algorithm
// - manages memory needs for genetic algorithm
// - deals with processing calls to CUDA callRK
// - exits when individuals converge on tolerance defined in Constants
double optimize(const cudaConstants* cConstants, GPUMem & gpuValues);

//Temp test function that is an assistant function to verify vectors
//Will attempt to find which sort an adult vector is in
//TODO: Should be removed, not used
void verifyVectorSort (const std::vector<Adult>& adults) {
    //Create flag bools that are used to try and find in what order each vector is sorted 
    bool rankSorted, distanceSorted, rankDistanceSorted;
    //Assume the vector is sorted in some ways 
    rankSorted = true;
    distanceSorted = true;
    rankDistanceSorted = true; 

    //go through the first 10 indicies of the vector (if it isn't empty)
    //if there is a pattern, it will report the potential sort
    if (adults.size() == 0)
    {
        //If its empty, report that
        std::cout << "empty vector"; 
    }
    else {
        for (int i = 0; i < 10; i++) {
            //Check to see if the vector is not in rank or rank-distance order
            //Since both rely on rank first, it is okay to check them at the same time
            if (adults[i].rank > adults[i+1].rank){
                //make rank and rank distance false, since it isn't sorted by any sort of rank
                rankSorted = rankDistanceSorted = false;
            }
            //see if rank is good, so check for distance for rank-distance
            else {
                //is only sorted by distance within a rank, so we need to see if the next adult is in the same rank
                if (adults[i].rank == adults[i+1].rank)
                {
                    //Check to see if the distances are not sorted
                    if (adults[i].distance < adults[i+1].distance)
                    {
                        //Array potentially in rank sort, but not rank-distance sort
                        rankDistanceSorted = false;
                    }
                    
                }
                
            }
            //Check to see if distances are not sorted correctly
            if (adults[i].distance < adults[i+1].distance)
            {
                distanceSorted = false; 
            }
            
        }

        //Not that the vector has been sorted through, report a probable sort, or report that no sort was identified
        if (rankDistanceSorted)
        {
            std::cout<<"rank-distance sorted";
        }
        else if (rankSorted)
        {
            std::cout<<"rank sorted";
        }
        else if (distanceSorted)
        {
            std::cout<<"distance sorted";
        }
        else {
            std::cout<<"no/unidentified sort";
        }
    }
}

//Temp test function to verify the status of the adult vectors
//Will report to the console the size of each vector and attempt to find out in what order they are sorted
//TODO: Should be removed, not used
void verifyVectors (const std::vector<Adult>& newAdults, const std::vector<Adult>& oldAdults, const std::vector<Adult>& allAdults, const std::string & areaUsed = "") {
    //Separate section for vector reporting
    std::cout << "\n_-_-_-_-_-_-_-_-VECTOR REPORTS-_-_-_-_-_-_-_-_\n";

    //Report where this is being used if reported
    if (areaUsed.size() > 0)
    {
        std::cout << "\nREPORT POINT: " << areaUsed << "\n";
    }
    
    //First start of reports for newAdults
    //Report size, set up reporting for sort
    std::cout << "\nnewAdults:\n\tsize: " << newAdults.size() << "\n\tsort status: ";

    //try to find sort status for newAdults
    verifyVectorSort(newAdults);

    //Next, report stats for oldAdults
    //Report size, set up reporting for sort
    std::cout << "\noldAdults:\n\tsize: " << oldAdults.size() << "\n\tsort status: ";

    //try to find sort status for newAdults
    verifyVectorSort(oldAdults);

    //Next, report stats for oldAdults
    //Report size, set up reporting for sort
    std::cout << "\nallAdults:\n\tsize: " << allAdults.size() << "\n\tsort status: ";

    //try to find sort status for newAdults
    verifyVectorSort(allAdults);

    std::cout << "\n\n_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_-_\n\n";
}

//-----------------------------------------------------------------------------------------------------------------------------
int main () {
    
    // display GPU properties and ensure we are using the right one
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    std::cout << "\n\nDevice Number: 0 \n";
    std::cout << "- Device name: " << prop.name << std::endl << std::endl;
    hipSetDevice(0);

    // Declare the genetic constants used, with file path being used to receive initial values
    cudaConstants * cConstants = new cudaConstants(); 

    //preallocates all the memory for the varaibles used by the GPU
    //also allows the GPU to access the marsLaunchCon without reloading it everytime
    GPUMem gpuValues;

    // Sets run0 seed, used to change seed between runs
    // Seed is set in cudaConstants: current time or passed in via config
    double zero_seed = cConstants->time_seed;
    // Perform the optimization with optimize function
    for (int run = 0; run < cConstants->run_count; run++) {
        // Adjust the time_seed so it is unique based on each run
        cConstants->time_seed = zero_seed + run*100;

        // Display contents of cConstants being used for this run and how many runs
        std::cout << *cConstants;
        std::cout << "\tPerforming run #" << run+1 << "\n\n";

        // pre-calculate a table of Earth's and Mars' position within possible mission time range
        // defined as global variable
        // accessed on the CPU when individuals are initialized
        launchCon = new PlanetInfo(cConstants, EARTH); 
        marsLaunchCon = new PlanetInfo(cConstants, MARS);
        //This ensures that we copy the correct size of marsCon to the GPU
        int marsConSize = getPlanetSize(cConstants);
        //initialize all values needed for GPU calculations
        gpuValues.initialize(cConstants, marsConSize, marsLaunchCon->getAllPositions());

        // Call optimize with the current parameters in cConstants
        optimize(cConstants, gpuValues);
        
        // Deallocate launchCon info for this run as it may be using a different time range in the next run
        delete launchCon; 
        delete marsLaunchCon;
        gpuValues.free();
    }
    // Now that the optimize function is done (assumed that optimize() also records it), deallocate memory of the cudaConstants
    delete cConstants;
    
    return 0;
}

//Returns true if top best_count adults within the oldAdults vector are within the tolerance
bool checkTolerance(std::vector<Adult>& oldAdults, const cudaConstants* cConstants) {
    //Sort the vector by rank distance to make sure the program checks the correct adult
    std::sort(oldAdults.begin(), oldAdults.end(), rankDistanceSort); 

    //The function needs to check if the best adult meets the convergence tolerance for each objective
    //Iterate through the objectives
    for (int i = 0; i < cConstants->missionObjectives.size(); i++) {

        if (cConstants->missionObjectives[i].goal < 0) {//Minimization
            //Check to see if the top best_count adults have met convergence for this parameter
            for (int j = 0; j < cConstants->best_count; j++) {

                //Check to see if the adult's parameter is larger than the convergence 
                if (oldAdults[j].getParameters(cConstants->missionObjectives[i]) > cConstants->missionObjectives[i].convergenceThreshold) {
                    //Return false as a parameter that needs to be minimized is larger than the convergence threshold
                    return false;
                }
            }
        }
        else if (cConstants->missionObjectives[i].goal > 0) {//Maximization
            //Check to see if the top best_count adults have met convergence for this parameter
            for (int j = 0; j < cConstants->best_count; j++) {
                //Check to see if the adult's parameter is smaller than the convergence 
                if (oldAdults[j].getParameters(cConstants->missionObjectives[i]) < cConstants->missionObjectives[i].convergenceThreshold) {
                    //Return false as a parameter that needs to be maximized is smaller than the convergence threshold
                    return false;
                }
            }
        }
        //No mission type was identified 
        else {
            std::cout << "\n_-_-_-_-_-_-_-_-_-Error Identifying Parameter Goal_-_-_-_-_-_-_-_-_-\n";
        }
    }

    //If the program reaches this spot, it means all of the adult's parameters have met the convergence threshold
    //  Otherwise, the function would have already returned false
    //  Thus, the adult has converged and it is appropriate to return true
    return true; 
}

//Function that will calculate distance and birthday values for a generation
void calculateGenerationValues (const std::vector<Adult> & allAdults, const std::vector<objective> & objectives, std::vector<double> & objectiveAvgValues, int & duplicateNum, double & minDist, double & avgDist, double & maxDist, const int & generation, double & avgAge, double & avgBirthday, int & oldestBirthday){
    //Reset the dist values
    minDist = 2; //Set the min dist to the maximum possible value, so that it will be changed
    avgDist = 0; 
    maxDist = 0; //Set the max dist to the min possible value, so that it is garunteed to be changed

    //Reset the average parameter values
    objectiveAvgValues.clear();
    //Prep the avg value vector to have an index for each objective
    for (int i = 0; i < objectives.size(); i++) {
        objectiveAvgValues.push_back(0.0); 
    }
    
    //Reset the age values
    avgAge = 0;
    avgBirthday = 0; 
    oldestBirthday = generation; //Set to generation, since it is the "newest" possible oldest birthday
    
    //For loop will iterate through the adult array to find the values needed
    for (int i = 0; i < allAdults.size(); i++) {
        //Check to see if this adult's distance is a new minimum
        if (allAdults[i].distance < minDist) {
            minDist = allAdults[i].distance; //Set the new min distance
        }

        //Check to see if this adult's distance is the new maximum
        if (allAdults[i].distance > maxDist) {
            maxDist = allAdults[i].distance; //Ser the new max distance
        }

        //Check to see if this adult's birthday is older than the current oldest
        if (allAdults[i].birthday < oldestBirthday) {
            oldestBirthday = allAdults[i].birthday; 
        }
           
        //Add to the avg distance
        avgDist += allAdults[i].distance;

        //Add the adult's parameter values to the necessary spot in the objective average value vector
        for (int j = 0; j < objectives.size(); j++) {
            objectiveAvgValues[j] += allAdults[i].getParameters(objectives[j]);
        }
        
        //Add to the avg age values
        //avgAge measures the age relative to the current generation (how many generations old), so it is generation minus the adults' birthdays 
        avgBirthday += allAdults[i].birthday;
        avgAge += (generation - allAdults[i].birthday);   
    }
    //Possible floating point roundoff error
    //  Shouldn't matter, as we don't ever compared averages to individuals
    //Divide the averages by the number of adults to get the averages
    avgDist /= allAdults.size();
    for (int i = 0; i < objectiveAvgValues.size(); i++) {
        objectiveAvgValues[i] /= allAdults.size();
    }
    avgAge /= allAdults.size(); 
    avgBirthday /= allAdults.size(); 
    
}

//Function that will facilitate the process of finding an optimal flight path
double optimize(const cudaConstants* cConstants, GPUMem & gpuValues) {
    // Not used, previously used for reporting computational performance
    double calcPerS = 0;

    time_t timeSeed = cConstants->time_seed;
    std::mt19937_64 rng(timeSeed); // This rng object is used for generating all random numbers in the genetic algorithm, passed in to functions that need it
    
    std::cout << "----------------------------------------------------------------------------------------------------" << std::endl;

    //Initialize the output object with the base folder location (..\Output_Files\)
    output genOutputs(cConstants);

    // input parameters for Runge Kutta process
    // Each parameter is the same for each thread on the GPU
    //double timeInitial = 0; // the starting time of the trip is always defined as zero   

    // Runge Kutta adaptive time step error tolerance
    // Not used now that callRK has been moved to ga_crossover
    //double absTol = cConstants->rk_tol; 

    // Initial genetic anneal scalar
    double currentAnneal = cConstants->anneal_initial;
    
    // Main set of parameters for Genetic Algorithm
    // contains all thread unique input parameters
    // The "children pool" of the current genertation
    std::vector<Adult> newAdults; 

    //Input parameters from the previous generation. Mixes with new children to determine new inputParameters
    // The "potential parent pool" of the current generation
    // DISCLAIMER - this is mentioned as the "parent pool", but it's actually the *POTENTIAL* parent pool. The survivor pool is what's used to generate the next generation. Survivors are taken from this, so it's more accurate to call this the "Potential Parent Pool"
    std::vector<Adult> oldAdults; 

    //the set of all old and new individuals
    std::vector<Adult> allAdults;

    // number of current generation
    int generation = 0;

    genOutputs.recordMarsData(cConstants,generation);
    genOutputs.recordEarthData(cConstants,generation);

    // Flag for finishing the genetic process
    // set by checkTolerance()
    bool convergence = false;

    //number of errors, specifically used for diagnostic recording
    //  couting all adults in the generation - includes oldAdults and newAdults that have nan values
    int numErrors = 0;
    int marsErrors = 0;

    //Initialize variables needed for distance, number of duplicate adults, and birthday reporting
    int duplicateNum = 0;
    double maxDistance, minDistance, avgDistance, avgAge, avgBirthday;
    int oldestBirthday;

    //Vector used to report the average parameter value for each objective
    std::vector<double> objectiveAvgValues; 

    //Creates the individuals needed for the 0th generation
    //Need to make children, then callRK, then make into adults (not currently doing that)
    //oldAdults goes into this function empty and comes out filled with num_individuals Adults
    //      these adults are either randomly generated or pulled from a file
    createFirstGeneration(oldAdults, cConstants, rng, generation, gpuValues); 


    // main gentic algorithm loop
    // - continues until checkTolerance returns true (specific number of individuals are within threshold)
    do {        
        // Genetic Crossover and mutation occur here
        //takes in oldAdults (the potential parents) and fills newAdults with descendants of the old adults
        //oldAdults is filled with the potential parents for a generation (num_individuals size) 
        //      after the run, oldAdults should remain the same
        //newAdults is empty and will be filled with the "grown" children generated in this method
        //std::cout << "\n\n_-_-_-_-_-_-_-_-_-TEST: PRE NEW GEN-_-_-_-_-_-_-_-_-_\n\n";
        newGeneration(oldAdults, newAdults, currentAnneal, generation, rng, cConstants, gpuValues);
        //Test function that will display the size and likely sort of each adult vector
        //      newAdults: should be N size & unsorted
        //      oldAdults: should be N size & rankDistance sorted
        //      allAdults: should be empty or ~2N size of previous generation's adults and rankDistance sorted depending on if its on the 0th generation or not
        //verifyVectors(newAdults, oldAdults, allAdults, "Post New Generation");

        //std::cout << "\n\n_-_-_-_-_-_-_-_-_-TEST: PRE PREP PARENTS-_-_-_-_-_-_-_-_-_\n\n";
        //fill oldAdults with the best adults from this generation and the previous generation so that the best parents can be selected (numErrors is for all adults in the generation - the oldAdults and the newAdults)
        //allAdults will be filled with the last generation's set of parents and their offspring when this starts (sorted by rankDistanceSort)
        //      by the end of this function, it will be filled with the new generation's set of parents and children sorted by rankDistanceSort
        //newAdults goes in with the "grown" children created in new generation (size num_individuals)
        //      by the end of the function, it is cleared
        //oldAdults goes in with the pool of potential parents that may have generated the newAdults
        //      by the end of the function, it is filled with the best num_individuals adults from allAdults (sorted by rankDistanceSort) 
        preparePotentialParents(allAdults, newAdults, oldAdults, numErrors, duplicateNum, cConstants, generation, currentAnneal, marsErrors);

        //Test function that will display the size and likely sort of each adult vector
        //      TODO: What should the states be? We should report them here to reference with the actual test
        //verifyVectors(newAdults, oldAdults, allAdults, "Post Prepare Parents");

        // Display a '.' to the terminal to show that a generation has been performed
        // This also serves to visually seperate the terminalDisplay() calls across generations 
        std::cout << '.';

        //Perform utitlity tasks (adjusting anneal and reporting data)
        //Calculate variables for birthdays and distances
        calculateGenerationValues(allAdults, cConstants->missionObjectives, objectiveAvgValues, duplicateNum, minDistance, avgDistance, maxDistance, generation, avgAge, avgBirthday, oldestBirthday);

        //Assumes oldAdults is in rankDistance order
        changeAnneal (oldAdults, cConstants, currentAnneal, generation);


        //std::cout << "\n\n_-_-_-_-_-_-_-_-_-TEST: PRE RECORD-_-_-_-_-_-_-_-_-_\n\n";
        //Print out necessary info for this generation
        genOutputs.printGeneration(cConstants, allAdults, objectiveAvgValues, generation, currentAnneal, numErrors, duplicateNum, minDistance, avgDistance, maxDistance, avgAge, generation-oldestBirthday, avgBirthday, oldestBirthday);

        //Test function that will display the size and likely sort of each adult vector
        //      TODO: What should the states be? We should report them here to reference with the actual test
        //verifyVectors(newAdults, oldAdults, allAdults, "End Of Generation");

        // Before replacing new adults, determine whether all are within tolerance
        // Determines when loop is finished
        //std::cout << "\n\n_-_-_-_-_-_-_-_-_-TEST: PRE CONVERGENCE CHECK-_-_-_-_-_-_-_-_-_\n\n";
        convergence = checkTolerance(oldAdults, cConstants);
        
        //Increment the generation counter
        ++generation;
    
        //Loop exits based on result of checkTolerance and if max_generations has been hit
    } while ( !convergence && generation < cConstants->max_generations);

    //Handle the final printing
    genOutputs.printFinalGen(cConstants, allAdults, convergence, generation, numErrors, duplicateNum, oldestBirthday); 

    return calcPerS;
}