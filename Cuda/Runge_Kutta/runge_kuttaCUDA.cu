#include "hip/hip_runtime.h"
#include <math.h>
#include <iostream>
#include <fstream> // for outputing to .csv file
#include <chrono>
#include <algorithm> // sort(), shuffle()
#include <random>

// Called by optimize() in optimization.cu
void callRK(const int numThreads, const int blockThreads, Individual *generation, double timeInitial, double stepSize, double absTol, double & calcPerS, const cudaConstants* cConstant) {
    
    hipEvent_t kernelStart, kernelEnd;
    hipEventCreate(&kernelStart);
    hipEventCreate(&kernelEnd);

    Individual *devGeneration; 
    double *devTimeInitial;
    double *devStepSize;
    double *devAbsTol;
    cudaConstants *devCConstant;

    // allocate memory for the parameters passed to the device
    hipMalloc((void**) &devGeneration, numThreads * sizeof(Individual));
    hipMalloc((void**) &devTimeInitial, sizeof(double));
    hipMalloc((void**) &devStepSize, sizeof(double));
    hipMalloc((void**) &devAbsTol, sizeof(double));
    hipMalloc((void**) &devCConstant, sizeof(cudaConstants));

    // copy values of parameters passed from host onto device
    hipMemcpy(devGeneration, generation, numThreads * sizeof(Individual), hipMemcpyHostToDevice);
    hipMemcpy(devTimeInitial, &timeInitial, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(devStepSize, &stepSize, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(devAbsTol, &absTol, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(devCConstant, cConstant, sizeof(cudaConstants), hipMemcpyHostToDevice);
    

    // GPU version of rk4Simple()
    hipEventRecord(kernelStart);
    rk4SimpleCUDA<<<(numThreads+blockThreads-1)/blockThreads,blockThreads>>>(devGeneration, devTimeInitial, devStepSize, devAbsTol, numThreads, devCConstant);
    hipEventRecord(kernelEnd);

    // copy the result of the kernel onto the host
    hipMemcpy(generation, devGeneration, numThreads * sizeof(Individual), hipMemcpyDeviceToHost);
    
    // free memory from device
    hipFree(devGeneration);
    hipFree(devTimeInitial);
    hipFree(devStepSize);
    hipFree(devAbsTol);
    hipFree(devCConstant);

    float kernelT;
    
    hipEventSynchronize(kernelEnd);

    hipEventElapsedTime(&kernelT, kernelStart, kernelEnd);
    
    calcPerS = numThreads / (kernelT / 1000.0); // how many times the Runge Kutta algorithm ran in the kernel per second
}

// seperate conditions are passed for each thread, but timeInitial, stepSize, and absTol are the same for every thread
__global__ void rk4SimpleCUDA(Individual *individuals, double *timeInitial, double *startStepSize, double *absTolInput, int n, const cudaConstants* cConstant) {
    int threadId = threadIdx.x + blockIdx.x * blockDim.x;
    if (threadId < n) {
        rkParameters<double> threadRKParameters = individuals[threadId].startParams; // get the parameters for this thread

        elements<double> curPos = threadRKParameters.y0; // start with the initial conditions of the spacecraft

        // storing copies of the input values
        double stepSize = *startStepSize;
        double absTol = *absTolInput;
        double curTime = *timeInitial;
        double startTime = *timeInitial;
        double curAccel = 0;

        thruster<double> thrust(cConstant);

        double massFuelSpent = 0; // mass of total fuel expended (kg) starts at 0

        bool coast; // to hold the result from calc_coast()

        elements<double> error; // holds output of previous value from rkCalc

        while (curTime < threadRKParameters.tripTime) {

            // Check the thruster type before performing calculations
            if (cConstant->thruster_type == thruster<double>::NO_THRUST) {
                coast = curAccel = 0;
            }
            else {
                coast = calc_coast(threadRKParameters.coeff, curTime, threadRKParameters.tripTime, thrust);
                curAccel = calc_accel(curPos.r, curPos.z, thrust, massFuelSpent, stepSize, coast, static_cast<double>(cConstant->wet_mass), cConstant);
            }

            // calculate k values and get new value of y
            rkCalc(curTime, threadRKParameters.tripTime, stepSize, curPos, threadRKParameters.coeff, curAccel, error); 

            curTime += stepSize; // update the current time in the simulation
            
            stepSize *= calc_scalingFactor(curPos-error,error,absTol, cConstant->doublePrecThresh); // Alter the step size for the next iteration

            // The step size cannot exceed the total time divided by 2 and cannot be smaller than the total time divided by 1000
            if (stepSize > (threadRKParameters.tripTime - startTime) / cConstant->min_numsteps) {
                stepSize = (threadRKParameters.tripTime - startTime) / cConstant->min_numsteps;
            }
            else if (stepSize < (threadRKParameters.tripTime - startTime) / cConstant->max_numsteps) {
                stepSize = (threadRKParameters.tripTime - startTime) / cConstant->max_numsteps;
            }
            
            if ( (curTime + stepSize) > threadRKParameters.tripTime) {
                stepSize = (threadRKParameters.tripTime - curTime); // shorten the last step to end exactly at time final
            }

            // if the spacecraft is within 0.5 au of the sun, the radial position of the spacecraft artificially increases to 1000, to force that path to not be used in the optimization.
            if ( sqrt(pow(curPos.r,2) + pow(curPos.z,2)) < 0.5) {
                //This is a bad result, needs to be set to be removed
                // Right after callRK we check for NaN in the elements, then reset individual
                individuals[threadId].finalPos.r = nan("");

                //Just to make sure invalidating posDiff/velDiff
                individuals[threadId].posDiff = nan("");
                individuals[threadId].velDiff = nan("");            

                return;
            }
        }

         // output to this thread's index
        individuals[threadId].finalPos = curPos;

        // Calculate new values for this thread
        individuals[threadId].getPosDiff(cConstant);
        individuals[threadId].getVelDiff(cConstant);
        individuals[threadId].getCost(cConstant);

        return;
    }
    return;
}

